#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ 
void calculate(int *result, int *count, int N)
{
	int idx = threadIdx.x;
	int idy = blockIdx.x;
	
	if(idx < N && idy < N){
	
		if(idx*idx + idy*idy > N*N){
			atomicAdd(result, 1);
		}
		
		atomicAdd(count, 1);
	}
}

int main(void)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int N = 1<<10;
	
	unsigned int threadsPerBlock  = N;
    unsigned int blockCount = N;
		
	int *result = 0;
	int count = N*N;
	
	hipEventRecord(start);
		
	hipMallocManaged(&result, sizeof(int));
	hipMallocManaged(&count, sizeof(int));
	
	calculate<<<blockCount, threadsPerBlock>>>(result, count, N);
	hipDeviceSynchronize();
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	result[0] = count - result[0];

	std::cout << "Result: " << result[0] << std::endl;
	std::cout << "Count: " << count << std::endl;
	std::cout << "Pi: " << (float)result[0]/count*4 << std::endl;
	std::cout << "Time: " << milliseconds << std::endl;
	

	hipFree(result);
  
	return 0;
}