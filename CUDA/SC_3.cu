#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <cstring>
#include <string>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_vector_types.h>

#include "openMP.hpp"
#include "CUDA_wrappers.hpp"
#include "common/image_helpers.hpp"

template <class T1, class T2>
void prepareImagePointers(	const char * const inputImageFileName,
							cv::Mat& inputImage, 
							T1** inputImageArray, 
							cv::Mat& outputImage,
							T2** outputImageArray, 
							const int outputImageType)
{
	using namespace std;
	using namespace cv;

	inputImage = imread(inputImageFileName, IMREAD_COLOR);

	if (inputImage.empty()) 
	{
		cerr << "Couldn't open input file." << endl;
		exit(1);
	}

	//allocate memory for the output
	outputImage.create(inputImage.rows, inputImage.cols, outputImageType);

	cvtColor(inputImage, inputImage, cv::COLOR_BGR2BGRA);

	*inputImageArray = (T1*)inputImage.ptr<char>(0);
	*outputImageArray  = (T2*)outputImage.ptr<char>(0); 
}




using namespace cv;
using namespace std;

int main( int argc, char** argv )
{
  using namespace std::chrono;

  if( argc != 2)
  {
    cout <<" Usage: convert_to_grayscale imagefile" << endl;
    return -1;
  }

  Mat image, imageGray;
  uchar4 *imageArray;
  unsigned char *imageGrayArray;

  prepareImagePointers(argv[1], image, &imageArray, imageGray, &imageGrayArray, CV_8UC1);

  int numRows = image.rows, numCols = image.cols;

  auto start = system_clock::now();
  RGBtoGrayscaleOpenMP(imageArray, imageGrayArray, numRows, numCols);
  auto duration = duration_cast<milliseconds>(system_clock::now() - start);
  cout<<"OpenMP time (ms):" << duration.count() << endl;

  memset(imageGrayArray, 0, sizeof(unsigned char)*numRows*numCols);  

  RGBtoGrayscaleCUDA(imageArray, imageGrayArray, numRows, numCols);

  return 0;
}