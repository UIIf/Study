#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string>

#include <hip/hip_runtime.h>
#include "SC_3.h"

using namespace std;


 
 
static unsigned short read_u16(FILE *fp);
static unsigned int   read_u32(FILE *fp);
static int            read_s32(FILE *fp);

static void write_u16(FILE *fp, unsigned short val);
static void write_u32(FILE *fp, unsigned int val);
static void write_s32(FILE *fp, int val);



__global__ 
void make_gray_scale(RGBQUAD **rgb,int height, int N)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
	
    int x = id/height;
    int y = id%height;

	if(id < N){
        rgb[x][y].grayScale = char(0.299*rgb[x][y].r + 0.587*rgb[x][y].g + 0.114*rgb[x][y].b);
	}
		
}

int main()
{
    BITMAPFILEHEADER header __attribute__((unused));
    BITMAPINFOHEADER bmiHeader;

    string eOfStr;
    string ending;

    RGBQUAD **rgb = read_bmp(header, bmiHeader, eOfStr,ending);

    cout<<"Read"<<endl;

    int N = bmiHeader.biHeight * bmiHeader.biWidth;

    int threadsCount = 512;
    int blockCount = N/threadsCount + 1;

    cout<<rgb[0][0].r<<' '<<rgb[0][0].g<<' '<<rgb[0][0].b<<endl;
    make_gray_scale<<<blockCount, threadsCount>>>(rgb, bmiHeader.biHeight, N);
    hipDeviceSynchronize();
    cout<<rgb[0][0].grayScale<<endl;
    cout<<"Gray"<<endl;

    write_bmp(rgb, header, bmiHeader, eOfStr,ending);

    cout<<"Write"<<endl;

    for(int i = 0; i < bmiHeader.biWidth; i++){
        hipFree(rgb[i]);
    }
    hipFree(rgb);

    return 0;
}
 
 

 
 