#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/transform_reduce.h>

struct inside_circle {
	__device__ unsigned int operator()(float2 p) const {
		return (((p.x-0.5)*(p.x-0.5)+(p.y-0.5)*(p.y-0.5))<0.25) ? 1 : 0;
	}
};

struct randomPoint {
	__device__ float2 operator() (const unsigned int n) {
		thrust::default_random_engine rnd;
		rnd.discard(2*n);
		return make_float2 (
			(float)rnd()/thrust::default_random_engine::max,
			(float)rnd()/thrust::default_random_engine::max);
	}
};

int main()
{	
	int N = 1<<20;
	thrust::device_vector<float2> d_random(N);
	thrust::counting_iterator<unsigned int> d_indexSequence(N);
	thrust::transform(d_indexSequence, d_indexSequence + N, d_random.begin(), randomPoint());
	
	
	int sum = thrust::transform_reduce(d_random.begin(), d_random.end(), inside_circle(), 0, thrust::plus<int>());

	
	/*for(int i = 0; i < N; i++)
		std::cout << d_inside[i] << " " << std::endl;*/
	
	std::cout << 4 * (float)sum/N << std::endl;
	
	
	
	/*hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int N = 1<<10;
	
	thrust::device_vector <int> calc(N*N);
	
	thrust::device_vector <float> x(N);
	thrust::device_vector <float> y(N);
	
	thrust::sequence (x.begin, x.end, 0, (float)1/N)
	thrust::sequence (y.begin, y.end, 0, (float)1/N)
	
	
	
	hipEventRecord(start);

	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	std::cout << "Result: " << result[0] << std::endl;
	std::cout << "Count: " << count[0] << std::endl;
	std::cout << "Pi: " << (float)result[0]/count[0]*4 << std::endl;
	std::cout << "Time: " << milliseconds << std::endl;*/
	
  
	return 0;
}