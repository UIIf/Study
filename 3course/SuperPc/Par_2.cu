#include <iostream>
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

// Kernel function to add the elements of two arrays
__global__ 
void setup_kernel(hiprandState *state, int N)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
	if(id < N)
		hiprand_init(1234, id, 0, &state[id]);
}

__global__ 
void generate_kernel(hiprandState *state, unsigned int *result, int N)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float x, y;
	
	if(id < N){
		hiprandState localState = state[id];
		x = hiprand_uniform(&localState);
		y = hiprand_uniform(&localState);
		
		state[id] = localState;
		
		result[id] = x*x + y*y <= 1;
	}
		
}

int main(void)
{	
	int N = 1<<20;
	
	unsigned int threadsPerBlock = 256;
    unsigned int blockCount = N/256 + 1;
		
	hiprandState *devStates;
	unsigned int *result;
		
	CUDA_CALL(hipMallocManaged(&result, N*sizeof(unsigned int)));
	CUDA_CALL(hipMallocManaged(&devStates, N*sizeof(hiprandState)));

  // Run kernel on 1M elements on the GPU
	setup_kernel<<<blockCount, threadsPerBlock>>>(devStates, N);
	hipDeviceSynchronize();
	
	generate_kernel<<<blockCount, threadsPerBlock>>>(devStates, result, N);
	hipDeviceSynchronize();

	float sum = 0;
	for (int i = 0; i < N; i++)
		sum += result[i] * 4;
	
	std::cout << "Sum: " << sum << std::endl;
	std::cout << "N: " << N << std::endl;
	std::cout << "BC: " << blockCount << std::endl;
	std::cout << "Sum/N: " << sum/(float)N << std::endl;

  // Free memory
	hipFree(devStates);
	hipFree(result);
  
  return 0;
}