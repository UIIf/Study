#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void trapezoidalIntegral(double* sum, double h, double begin) {
	double x_1 = begin + i * h, x_2 = begin + (i + 1) * h;
    double y_1 = 2*sqrt( (float) 1 - x_1*x_1);
    double y_2 = 2*sqrt( (float) 1 - x_2*x_2);

    sum[i] = (y_1 + y_2)/2*h;
}

__global__ void reduce(double* res) {
	unsigned int tid = threadIdx.x;

	for (size_t k = 1; k < blockDim.x; k *= 2) {
		unsigned int index = 2 * k * tid;

		if (index < blockDim.x) res[index] += res[index + k];
		__syncthreads();
	}
}

int main() {
	const double a = 0, b = 15;
	const int n = 1000;
	double h = (b - a) / n, result = 0;
	double* mas_c;

	hipMalloc((void**)&mas_c, n * sizeof(double));

	trapezoidalIntegral << <1, n >> > (mas_c, h, a);
	reduce << <1, n >> > (mas_c);

	hipMemcpy(&result, mas_c, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(mas_c);
	printf("Result = %f", result);

	return 0;
} // nvcc trapezoidal.cu -o trapezoidal